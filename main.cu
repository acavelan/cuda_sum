
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <chrono>

__global__ void cuda_vecAdd(float *v1, float *v2, float *v3, int offset)
{
	int i = offset + blockIdx.x * blockDim.x + threadIdx.x;
	v3[i] = v1[i] + v2[i];
}

int main(void)
{
	typedef std::chrono::high_resolution_clock Clock;
	typedef std::chrono::time_point<Clock> TimePoint;
	typedef std::chrono::duration<float> Time;

	constexpr int n = 256*1000000;

	constexpr int nStreams = 32;
	constexpr int blockSize = 256;
	constexpr int streamSize = n / nStreams;
	constexpr int gridSize = streamSize / blockSize;

	printf("%d %d\n", streamSize, gridSize);

	TimePoint tstart, tstop;

	hipStream_t streams[nStreams];

	float *v1 = nullptr, *v2 = nullptr, *v3 = nullptr;
	float *dv1 = nullptr, *dv2 = nullptr, *dv3 = nullptr;

	for(int i=0; i<nStreams; i++)
		hipStreamCreate(&streams[i]);

	hipHostMalloc((void**)&v1, sizeof(float) * n, hipHostMallocDefault);
	hipHostMalloc((void**)&v2, sizeof(float) * n, hipHostMallocDefault);
	hipHostMalloc((void**)&v3, sizeof(float) * n, hipHostMallocDefault);

	hipMalloc((void**)&dv1, sizeof(float) * n);
	hipMalloc((void**)&dv2, sizeof(float) * n);
	hipMalloc((void**)&dv3, sizeof(float) * n);

	for(int i=0; i<n; i++)
	{
		v1[i] = (float)i;
		v2[i] = (float)-i+1;
	}

	for(int i=0; i<n; i++)
		v3[i] = 0.0f;

	tstart = Clock::now();

	for(int i=0; i<nStreams; i++)
	{
		int offset = i * streamSize;

		hipMemcpyAsync(&dv1[offset], &v1[offset], sizeof(float) * streamSize, hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(&dv2[offset], &v2[offset], sizeof(float) * streamSize, hipMemcpyHostToDevice, streams[i]);

		cuda_vecAdd<<<gridSize, blockSize, 0, streams[i]>>>(dv1, dv2, dv3, offset);

		hipMemcpyAsync(&v3[offset], &dv3[offset], sizeof(float) * streamSize, hipMemcpyDeviceToHost, streams[i]);
	}
	
	for(int i=0; i<nStreams; i++)
		hipStreamSynchronize(streams[i]);
	
	tstop = Clock::now();

	float elaspedTimeGPU = std::chrono::duration_cast<Time>(tstop-tstart).count();

	float checkGPU = 0.0f;
	for(int i=0; i<n; i++)
		checkGPU += v3[i];

	printf("CheckGPU: %f -- TimeGPU: %fms\n", checkGPU, elaspedTimeGPU);

	hipFree(dv1);
	hipFree(dv2);
	hipFree(dv3);

	hipHostFree(v1);
	hipHostFree(v2);
	hipHostFree(v3);

	for(int i=0; i<nStreams; i++)
		hipStreamDestroy(streams[i]);

	return 0;
}